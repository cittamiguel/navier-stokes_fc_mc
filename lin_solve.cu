#include "hip/hip_runtime.h"
#include <stddef.h>
#include <hip/hip_runtime.h> // Core CUDA runtime APIs
#include <> // For thread/block intrinsics
extern "C" {
#include "solver.h"
}

__global__ void lin_solve_rb_step_kernel(grid_color color,
                              unsigned int n,
                              float a,
                              float c,
                              const float * __restrict__ same0,
                              const float * __restrict__ neigh,
                              float * __restrict__ same)
{
    unsigned int width = (n + 2) / 2;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y + 1;

    if (y > n) return;

    int shift = ((y % 2 == 0) ^ (color == RED)) ? 1 : -1;
    int start = (y % 2 == 0) == (color == RED) ? 0 : 1;

    for (unsigned int x = threadIdx.x + start; x < width - (1 - start); x += blockDim.x) {
        int index = y * width + x;

        same[index] = (same0[index] + a * (
            neigh[index - width] +
            neigh[index] +
            neigh[index + shift] +
            neigh[index + width])) / c;
    }
}

void lin_solve(unsigned int n, boundary b,
                   float * x, const float * x0,
                   float a, float c)
{
    unsigned int color_size = (n + 2) * ((n + 2) / 2);
    float *d_x, *d_x0;
    size_t total_size = 2 * color_size * sizeof(float);

    hipMalloc((void**)&d_x, total_size);
    hipMalloc((void**)&d_x0, total_size);

    hipMemcpy(d_x, x, total_size, hipMemcpyHostToDevice);
    hipMemcpy(d_x0, x0, total_size, hipMemcpyHostToDevice);

    float * red = d_x;
    float * blk = d_x + color_size;
    const float * red0 = d_x0;
    const float * blk0 = d_x0 + color_size;

    dim3 blockDim(32, 8);  // tunable
    dim3 gridDim(1, (n + blockDim.y - 1) / blockDim.y);

    for (unsigned int k = 0; k < 20; ++k) {
        lin_solve_rb_step_kernel<<<gridDim, blockDim>>>(RED, n, a, c, red0, blk, red);
        hipDeviceSynchronize();

        lin_solve_rb_step_kernel<<<gridDim, blockDim>>>(BLACK, n, a, c, blk0, red, blk);
        hipDeviceSynchronize();
        
        set_bnd(n, b, x);
    }

    hipMemcpy(x, d_x, total_size, hipMemcpyDeviceToHost);
    hipFree(d_x);
    hipFree(d_x0);
}